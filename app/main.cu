#include <hip/hip_runtime.h>
#include "foo_api.h"

void run_cuda(float* a, float* b, float* c, int n) {
    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));
    hipMalloc((void **)&d_c, n * sizeof(float));

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipMemcpyAsync(d_a, a, n * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b, n * sizeof(float), hipMemcpyHostToDevice, stream);

    // launch library kernel
    launch_external(d_a, d_b, d_c, n, stream);

    hipMemcpyAsync(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
}